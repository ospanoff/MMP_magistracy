#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdlib>
#include <cstdio>
#include <ctime>

#include <hip/hip_runtime.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>

#define  Max(a, b) ((a)>(b)?(a):(b))

#define CUDA_SAFE_CALL(call)\
do {\
    hipError_t err = call;\
    if (hipSuccess != err) {\
        printf("Cuda error in file '%s' in line %i: %s\n",\
        __FILE__, __LINE__, hipGetErrorString(err));\
        exit(1);\
    }\
} while (false)\

FILE *in;
int TRACE = 0;
double EPS;
int     M, N, K, ITMAX;
double  MAXEPS = 0.01;

double *A, *A_GPU;
double *diff;
thrust::device_ptr<double> diff_dev;

#define A(i,j,k) A[((i)*N+(j))*K+(k)]
#define a(i,j,k) a[((i)*nn+(j))*kk+(k)]
#define diff(i,j,k) diff[((i)*nn+(j))*kk+(k)]
#define a2(i,j,k) a2[((i)*n2+(j))*k2+(k)]

double solution(int i, int j, int k)
{
    double x = 10.*i / (M - 1), y = 10.*j / (N - 1), z = 10.*k / (K - 1);
    return 2.*x*x - y*y - z*z;
}

double jac(double *a, int mm, int nn, int kk, int itmax, double maxeps);

int main(int an, char **as)
{
    int i, j, k;
    in = fopen("data3.in", "r");
    if (in == NULL) { printf("Can not open 'data3.in' "); exit(1); }
    i = fscanf(in, "%d %d %d %d %d", &M, &N, &K, &ITMAX, &TRACE);
    if (i < 4)
    {
        printf("Wrong 'data3.in' (M N K ITMAX TRACE)");
        exit(2);
    }

    A = (double*) malloc(M*N*K * sizeof(double));

    for (i = 0; i <= M - 1; i++)
        for (j = 0; j <= N - 1; j++)
            for (k = 0; k <= K - 1; k++) {
                if (i == 0 || i == M - 1 || j == 0 || j == N - 1 || k == 0 || k == K - 1)
                    A(i, j, k) = solution(i, j, k);
                else 
                    A(i, j, k) = 0.;
            }

    CUDA_SAFE_CALL(hipMalloc(&A_GPU, M*N*K * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(&diff, M*N*K * sizeof(double)));
    CUDA_SAFE_CALL(hipMemset(diff, 0, M*N*K * sizeof(double)));
    diff_dev = thrust::device_pointer_cast<double>(diff);

    CUDA_SAFE_CALL(hipMemcpy(A_GPU, A, M*N*K * sizeof(double), hipMemcpyHostToDevice));
    clock_t t = clock();

    EPS = jac(A_GPU, M, N, K, ITMAX, MAXEPS);

    t = clock() - t;
    double elapsed = 1.0 * t / CLOCKS_PER_SEC;
    printf("%dx%dx%d x %d\t<", M, N, K, ITMAX);
    printf("%3.5f s.>\teps=%.4g\n", elapsed, EPS);

    CUDA_SAFE_CALL(hipMemcpy(A, A_GPU, M*N*K * sizeof(double), hipMemcpyDeviceToHost));
    
    if (TRACE)
    {
        EPS = 0.;
        for (i = 0; i <= M - 1; i++)
            for (j = 0; j <= N - 1; j++)
                for (k = 0; k <= K - 1; k++)
                    EPS = Max(fabs(A(i, j, k) - solution(i, j, k)), EPS);
        printf("delta=%.4g\n", EPS);
    }

    free(A);
    CUDA_SAFE_CALL(hipFree(A_GPU));
    CUDA_SAFE_CALL(hipFree(diff));
    return 0;
}

__global__
void jac_kernel(double *a, int mm, int nn, int kk, double *diff) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.z * blockDim.z + threadIdx.z;
    if (i >= 1 && i <= mm - 2 && j >= 1 && j <= nn - 2 && k >= 1 && k <= kk - 2) {
        double tmp = (a(i - 1, j, k) + a(i + 1, j, k) + a(i, j - 1, k) + a(i, j + 1, k)
                      + a(i, j, k - 1) + a(i, j, k + 1)) / 6.;
        diff(i, j, k) = fabsf(a(i, j, k) - tmp);
        a(i, j, k) = tmp;
    }
}

void run_jac_kernel(double *a, int mm, int nn, int kk, double *diff) {
    dim3 gridDim = dim3((kk + 31) / 32, (nn + 31) / 32, mm);
    dim3 blockDim = dim3(32, 32, 1);
    jac_kernel<<<gridDim, blockDim>>>(a, mm, nn, kk, diff);
    CUDA_SAFE_CALL(hipGetLastError());
}

__global__
void jac_kernel_inner1(double *a, int mm, int nn, int kk, double *a2, int n2, int k2) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.z * blockDim.z + threadIdx.z;
    int mm_ = mm - 2 * (1 - mm % 2);
    int nn_ = nn - 2 * (1 - nn % 2);
    int kk_ = kk - 2 * (1 - kk % 2);
    if (i <= mm_ - 1 && j <= nn_ - 1 && k <= kk_ - 1) {
        if (i % 2 == 0 && j % 2 == 0 && k % 2 == 0) {
            a2(i / 2, j / 2, k / 2) = a(i, j, k);
        }
    } else if (i <= mm - 1 && j <= nn - 1 && k <= kk - 1) {
        if (i == mm - 1 || j == nn - 1 || k == kk - 1) {
            a2(i / 2, j / 2, k / 2) = a(i, j, k);
        }
    }
}

void run_jac_kernel_inner1(double *a, int mm, int nn, int kk, double *a2, int n2, int k2) {
    dim3 gridDim = dim3((kk + 31) / 32, (nn + 31) / 32, mm);
    dim3 blockDim = dim3(32, 32, 1);
    jac_kernel_inner1<<<gridDim, blockDim>>>(a, mm, nn, kk, a2, n2, k2);
    CUDA_SAFE_CALL(hipGetLastError());
}

__global__
void jac_kernel_inner2(double *a, int mm, int nn, int kk, double *a2, int n2, int k2) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.z * blockDim.z + threadIdx.z;
    if (i >= 1 && i <= mm - 2 && j >= 1 && j <= nn - 2 && k >= 1 && k <= kk - 2) {
        a(i, j, k) = (
                             a2(i / 2, j / 2, k / 2) +
                             a2(i / 2, j / 2, k / 2 + k % 2) +
                             a2(i / 2, j / 2 + j % 2, k / 2) +
                             a2(i / 2, j / 2 + j % 2, k / 2 + k % 2) +
                             a2(i / 2 + i % 2, j / 2, k / 2) +
                             a2(i / 2 + i % 2, j / 2, k / 2 + k % 2) +
                             a2(i / 2 + i % 2, j / 2 + j % 2, k / 2) +
                             a2(i / 2 + i % 2, j / 2 + j % 2, k / 2 + k % 2)
                     ) / 8.;
    }
}

void run_jac_kernel_inner2(double *a, int mm, int nn, int kk, double *a2, int n2, int k2) {
    dim3 gridDim = dim3((kk + 31) / 32, (nn + 31) / 32, mm);
    dim3 blockDim = dim3(32, 32, 1);
    jac_kernel_inner2<<<gridDim, blockDim>>>(a, mm, nn, kk, a2, n2, k2);
    CUDA_SAFE_CALL(hipGetLastError());
}


double jac(double *a_gpu, int mm, int nn, int kk, int itmax, double maxeps)
{
    int it, vecSize = mm*nn*kk;
    double eps;

    if (mm > 31 && nn > 31) {
        int m2 = (mm + 1) / 2, n2 = (nn + 1) / 2, k2 = (kk + 1) / 2;

        double *a2_gpu;
        int vecSizeInner = m2*n2*k2;

        CUDA_SAFE_CALL(hipMalloc(&a2_gpu, vecSizeInner * sizeof(double)));

        run_jac_kernel_inner1(a_gpu, mm, nn, kk, a2_gpu, n2, k2);

        eps = jac(a2_gpu, m2, n2, k2, itmax * 2, maxeps);

        run_jac_kernel_inner2(a_gpu, mm, nn, kk, a2_gpu, n2, k2);

        CUDA_SAFE_CALL(hipFree(a2_gpu));
    }

    for (it = 1; it <= itmax; it++) {
        run_jac_kernel(a_gpu, mm, nn, kk, diff);
        eps = thrust::reduce(
                diff_dev, diff_dev + vecSize, 0.0f, thrust::maximum<double>()
        );

        if (TRACE && it % TRACE == 0)
            printf("IT=%d eps=%.4g\n", it, eps);
        if (eps < maxeps) 
            break;
    }
    return eps;
}


